#include "hip/hip_runtime.h"
#include <stdio.h>
#include <conio.h>
#include <Windows.h>
#include <Ole2.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <gl/GL.h>
#include <gl/GLU.h>
#include <GL/glut.h>

// Preprocessor definitions for width and height of color and depth streams
#define CWIDTH 640
#define CHEIGHT 480
#define DWIDTH 640
#define DHEIGHT 480

/* Kernel to map color to the depth image space coordinates. 
   the inputs are color-space coordinates, buffer to fill up
   the mapped color */
__global__
void colordepthmap(BYTE* colorFrame, BYTE* mappedColor, LONG* colorCoordinates, int CtoDdiv)
{
  int x = blockIdx.x*blockDim.x + threadIdx.x;
	int y = blockIdx.y*blockDim.y + threadIdx.y;

	if (x >= CWIDTH || y >= CHEIGHT)
    {
        return;
    }

	int depthIndex = (y/CtoDdiv) * DWIDTH + (x/CtoDdiv);

	// extracting color coordinates mapped from the depth position (depthIndex)
	LONG colorForDepthX = colorCoordinates[depthIndex*2];
	LONG colorForDepthY = colorCoordinates[depthIndex*2 + 1];

	// check if the color coordinates lie within the range of the color map
	if(colorForDepthX >= 0 && colorForDepthX < CWIDTH && colorForDepthY >= 0 && colorForDepthY < CHEIGHT)
	{
		// calculate index in the color image
		int colorIndex = colorForDepthY * (CWIDTH*4) + (colorForDepthX*4);
		int m_ColorIndex = (y/CtoDdiv)*(DWIDTH*4)+(x/CtoDdiv)*4;

		mappedColor[m_ColorIndex] = colorFrame[colorIndex];
		mappedColor[m_ColorIndex+1] = colorFrame[colorIndex+1];
		mappedColor[m_ColorIndex+2] = colorFrame[colorIndex+2];
		mappedColor[m_ColorIndex+3] = colorFrame[colorIndex+3];
	}
}


// Function to initiate the kernel call
void mappingFunc(BYTE* h_colorFrame, BYTE* h_mappedColor, LONG* h_colorCoordinates, int CtoDiv)
{
	// Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

	/***** Allocating memory on the device/GPU *****/
	BYTE* d_colorFrame = NULL;
	BYTE* d_mappedColor = NULL;
	LONG* d_colorCoordinates = NULL;

	// Allocating memory for the color frame
	err = hipMalloc(&d_colorFrame, sizeof(BYTE)* 4 * CHEIGHT * CWIDTH);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Allocating memory for mapped color
	err = hipMalloc(&d_mappedColor, sizeof(BYTE)* 4 * DHEIGHT * DWIDTH);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Allocating memory for color coordinates
	err = hipMalloc(&d_colorCoordinates, sizeof(LONG)* 2 * DHEIGHT * DWIDTH);

	if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device memory for image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }




	/***** Copying data from host to device *****/
	err = hipMemcpy(d_colorFrame, h_colorFrame, sizeof(BYTE)* 4 * CHEIGHT * CWIDTH, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_colorCoordinates, h_colorCoordinates, sizeof(LONG)* 2 * DHEIGHT * DWIDTH, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy image from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }




	/***** Launching the Kernel code *****/
	dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid((CWIDTH+ 16-1)/16, (CHEIGHT+ 16-1)/16, 1);
	colordepthmap<<<blocksPerGrid, threadsPerBlock>>>(d_colorFrame, 
	                                                  d_mappedColor, 
													                          d_colorCoordinates, 
													                          CtoDiv);
	err = hipGetLastError(); hipDeviceSynchronize();



	/***** Copying data back from device to host *****/
	err = hipMemcpy(h_mappedColor, d_mappedColor, sizeof(BYTE)* 4 * DHEIGHT * DWIDTH, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to copy image from device to host (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }



	/***** Freeing the memory allocated in the device *****/
	err = hipFree(d_mappedColor);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device mapped image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipFree(d_colorFrame);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device mapped image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipFree(d_colorCoordinates);

    if (err != hipSuccess)
    {
        fprintf(stderr, "Failed to free device mapped image (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}
